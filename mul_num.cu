#include "hip/hip_runtime.h"
//MULTIPLY INTEGER WITH ARRAY
#include <stdio.h>
#include <stdlib.h>

#define N 100

__global__ void MatMulInt(int *a, int b, int *c,int ROW, int COLUMNS){
    
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = iy * COLUMNS + ix;

    if (ix < ROW && iy < COLUMNS)
    {
        c[idx] = a[idx] * b ;
    }
}

int main(){
    int ROW, COLUMNS;
    int b;

    //GET SIZES OF ARRAYS
    printf("please give rows of the array: ");
    scanf("%d",&ROW);
    
    printf("please give columns of the array ");
    scanf("%d",&COLUMNS);

    printf("please give integer you want to multiply ");
    scanf("%d",&b);
    
   
    int a[ROW][COLUMNS];

    int c[ROW][COLUMNS];



    for(int i=0;i<ROW;i++)
      for(int j=0;j<COLUMNS;j++)
      {
        a[i][j]=rand()%5;
      }

    //DEFINE POINTERS FOR GPU
    int *dev_a,  *dev_c;

    //DEFINE SIZE
    int size = ((ROW* COLUMNS)*sizeof(int));

    //ALLOCATE MEMORY FOR POINTERS
    hipMalloc((void**)&dev_a, size);
    hipMalloc((void**)&dev_c, size);

    //TRANSFER TO HOST
    hipMemcpy(dev_a, a, size ,hipMemcpyHostToDevice);

    hipMemcpy(dev_c, c, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 grid((ROW + threadsPerBlock.x - 1) / threadsPerBlock.x, (COLUMNS + threadsPerBlock.y - 1) / threadsPerBlock.y);

//    dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);
//    dim3 dimGrid( ceil(int(N)/int(threadsPerBlock.x)), ceil(int(N)/int(threadsPerBlock.y)) );


    clock_t start = clock();
    MatMulInt<<<grid,threadsPerBlock>>>(dev_a, b, dev_c, ROW, COLUMNS);
    clock_t end = clock();



    //RETURN RESULT TO DEVICE
    hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);



    hipFree(dev_a); 

    hipFree(dev_c);


//    printf("\n");
//
//    for(int i=0;i<ROW;i++)
//    {
//      for(int j=0;j<COLUMNS;j++)
//      {
//        printf("%d ",a[i][j]);
//      }
//      printf("\n");
//    }
//
//
//    printf("\n\n\n");
//
//
//  
//    printf("%d ",b);
//
// 
//    printf("\n\n\n");
//
//
//
//
//    for(int i=0;i<ROW;i++)
//    {
//      for(int j=0;j<COLUMNS;j++)
//      {
//        printf("%d ",c[i][j]);
//
//      }
//      printf("\n");
//    }

    float seconds = (float)(end - start) / CLOCKS_PER_SEC;
    printf("Xronos GPU %f\n\n",seconds);


    clock_t start_cpu = clock();
     for(int i=0;i<ROW;i++)
    {
        for (int j=0;j<COLUMNS;j++)
        {
            c[i][j]= a[i][j]  *b;
        }
    }

    clock_t end_cpu = clock();


    float secondss = (float)(end_cpu - start_cpu) / CLOCKS_PER_SEC;
    printf("Xronos CPU %f",secondss);






    return 0;
}
