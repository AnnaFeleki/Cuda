#include "hip/hip_runtime.h"
//MATRIX MULTIPLICATION
#include<stdio.h>
#include<stdlib.h>
#include <hip/hip_runtime.h>

__global__ void matrixMul(int *a, int *b, int *c, int ROW, int COLUMNS, int temp)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if( col < COLUMNS && row < ROW) 
    {
        for(int i = 0; i < temp; i++) 
        {
            sum += a[row * temp + i] * b[i * COLUMNS + col];
        }
        c[row * COLUMNS + col] = sum;
    }
    

}

int main()
{
   
    int ROW, COLUMNS;
    int temp;
    

    //GET SIZES OF ARRAYS
    printf("please give rows of A: ");
    scanf("%d",&ROW);
    
    printf("please give columns of A ");
    scanf("%d",&temp);

    printf("please give columns of B ");
    scanf("%d",&COLUMNS);
    
   
    int a[ROW][temp];
    int b[temp][COLUMNS];
    int c[ROW][COLUMNS];



    //FILL WITH RANDOM VALUES
    for(int i=0;i<ROW;i++)
      for(int j=0;j<temp;j++)
      {
        a[i][j]=rand()%5;
      }

     for(int i=0;i<temp;i++)
      for(int j=0;j<COLUMNS;j++)
      {
        b[i][j]=rand()%5;
      }


    //DEFINE POINTERS FOR GPU
    int *dev_a, *dev_b, *dev_c;


    //DEFINE SIZE
    int size_a = ((ROW * temp)*sizeof(int));
    int size_b = ((temp * COLUMNS)*sizeof(int));
    int size_c = ((ROW * COLUMNS)*sizeof(int));

    //ALLOCATE MEMORY FOR POINTERS
    hipMalloc((void**)&dev_a, size_a);
    hipMalloc((void**)&dev_b, size_b);
    hipMalloc((void**)&dev_c, size_c);

    //TRANSFER TO HOST
    hipMemcpy(dev_a, a, size_a ,hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size_b, hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, size_c, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 grid((ROW + threadsPerBlock.x - 1) / threadsPerBlock.x, (COLUMNS + threadsPerBlock.y - 1) / threadsPerBlock.y);

//    dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);
//    dim3 dimGrid( ceil(int(N)/int(threadsPerBlock.x)), ceil(int(N)/int(threadsPerBlock.y)) );


    clock_t start = clock();
    matrixMul<<<grid,threadsPerBlock>>>(dev_a, dev_b, dev_c, ROW, COLUMNS, temp);
    clock_t end = clock();


    //RETURN RESULT TO DEVICE
    hipMemcpy(c, dev_c, size_c, hipMemcpyDeviceToHost);



    hipFree(dev_a); 
    hipFree(dev_b); 
    hipFree(dev_c);


    printf("\n");
//
//    for(int i=0;i<ROW;i++)
//    {
//      for(int j=0;j<temp;j++)
//      {
//        printf("%d ",a[i][j]);
//      }
//      printf("\n");
//    }
//
//
//    printf("\n\n\n");
//
//
//    for(int i=0;i<temp;i++)
//    {
//      for(int j=0;j<COLUMNS;j++)
//      {
//        printf("%d ",b[i][j]);
//      }
//      printf("\n");
//    }
//
//    printf("\n\n\n");
//
//
//
//
//    for(int i=0;i<ROW;i++)
//    {
//      for(int j=0;j<COLUMNS;j++)
//      {
//        printf("%d ",c[i][j]);
//
//      }
//      printf("\n");
//    }
//
    
    float seconds = (float)(end - start) / CLOCKS_PER_SEC;
    printf("Xronos GPU %f\n\n",seconds);


    clock_t start_cpu = clock();
     //Carrying out matrix multiplication operation
    int i, j, k; 
        for (i = 0; i < ROW; i++) 
        { 
            for (j = 0; j < COLUMNS; j++) 
            { 
                c[i][j] = 0; 
                for (k = 0; k < temp; k++) 
                    c[i][j] += a[i][k]*b[k][j]; 
            } 
        } 

    clock_t end_cpu = clock();


    float secondss = (float)(end_cpu - start_cpu) / CLOCKS_PER_SEC;
    printf("Xronos CPU %f",secondss);



    return 0;
}
