#include "hip/hip_runtime.h"
//ADD TWO MATRICES
#include <stdio.h>
#include <stdlib.h>


__global__ void MatAdd(int *a, int *b, int *c, int ROW, int COLUMNS){
    
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = iy * COLUMNS + ix;

    if (ix < ROW && iy < COLUMNS)
    {
        c[idx] = a[idx] + b[idx];
    }
}



int main(){

    //GET SIZES OF ARRAYS
    int ROW, COLUMNS;
    printf("please give rows of the array: ");
    scanf("%d",&ROW);
    
    printf("please give columns of the array ");
    scanf("%d",&COLUMNS);
    
   
    int a[ROW][COLUMNS];
    int b[ROW][COLUMNS];
    int c[ROW][COLUMNS];


    //FILL WITH RANDOM VALUES
    for(int i=0;i<ROW;i++)
      for(int j=0;j<COLUMNS;j++)
      {
        a[i][j]=rand()%5;
        b[i][j]=rand()%5;
      }

    //DEFINE POINTERS FOR GPU
    int *dev_a, *dev_b, *dev_c;

    //DEFINE SIZE
    int size = ((ROW* COLUMNS)*sizeof(int));

    //ALLOCATE MEMORY FOR POINTERS
    hipMalloc((void**)&dev_a, size);
    hipMalloc((void**)&dev_b, size);
    hipMalloc((void**)&dev_c, size);

    //TRANSFER TO HOST
    hipMemcpy(dev_a, a, size ,hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, size, hipMemcpyHostToDevice);



    
    /*Do something*/
    
    

    dim3 threadsPerBlock(16, 16);
    dim3 grid((ROW + threadsPerBlock.x - 1) / threadsPerBlock.x, (COLUMNS + threadsPerBlock.y - 1) / threadsPerBlock.y);

    

//    dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);
//    dim3 dimGrid( ceil(int(N)/int(threadsPerBlock.x)), ceil(int(N)/int(threadsPerBlock.y)) );



    clock_t start = clock();
    MatAdd<<<grid,threadsPerBlock>>>(dev_a, dev_b, dev_c, ROW, COLUMNS);
    clock_t end = clock();


    //RETURN RESULT TO DEVICE
    hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);



    hipFree(dev_a); 
    hipFree(dev_b); 
    hipFree(dev_c);


    printf("\n");
//
//    for(int i=0;i<ROW;i++)
//    {
//      for(int j=0;j<COLUMNS;j++)
//      {
//        printf("%d ",a[i][j]);
//      }
//      printf("\n");
//    }
//
//
//    printf("\n\n\n");
//
//
//    for(int i=0;i<ROW;i++)  
//    {
//      for(int j=0;j<COLUMNS;j++)
//      {
//        printf("%d ",b[i][j]);
//
//      }
//      printf("\n");
//    }
//
//    printf("\n\n\n");
//
//
//
//
//    for(int i=0;i<ROW;i++)
//    {
//      for(int j=0;j<COLUMNS;j++)
//      {
//        printf("%d ",c[i][j]);
//
//      }
//      printf("\n");
//    }
//
//

    float seconds = (float)(end - start) / CLOCKS_PER_SEC;
    printf("Xronos GPU %f\n\n",seconds);


    clock_t start_cpu = clock();
     for(int i=0;i<ROW;i++)
    {
        for (int j=0;j<COLUMNS;j++)
        {
            c[i][j]= a[i][j]  + b[i][j];
        }
    }

    clock_t end_cpu = clock();


    float secondss = (float)(end_cpu - start_cpu) / CLOCKS_PER_SEC;
    printf("Xronos CPU %f",secondss);


    

    return 0;
}
